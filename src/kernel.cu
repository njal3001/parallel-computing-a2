#include "hip/hip_runtime.h"
/*
	CS3210 Assignment 2
	CUDA Virus Scanning

	Most of your CUDA code should go in here.

	Feel free to change any code in the skeleton, as long as you conform
	to the input and output formats specified in the assignment pdf.

	If you rename this file or add new files, remember to modify the
	Makefile! Just make sure (haha) that the default target still builds
	your program, and you don't rename the program (`scanner`).

	The skeleton demonstrates how asnychronous kernel launches can be
	done; it is up to you to decide (and implement!) the parallelisation
	paradigm for the kernel. The provided implementation is not great,
	since it launches one kernel per file+signature combination (a lot!).
	You should try to do more work per kernel in your implementation.

	You can launch as many kernels as you want; if any preprocessing is
	needed for your algorithm of choice, you can also do that on the GPU
	by running different kernels.

	'defs.h' contains the definitions of the structs containing the input
	and signature data parsed by the provided skeleton code; there should
	be no need to change it, but you can if you want to.

	'common.cpp' contains the aforementioned parsing for the input files.
	The input files are already efficiently read with mmap(), so there
	should be little to no gain trying to optimise that portion of the
	skeleton.

	Remember: print any debugging statements to STDERR!
*/

#include <vector>
#include <sstream>
#include <iomanip>

#include "defs.h"


// Kernel to test if 2 strings are equal
__device__ bool equal(const char* a, const char* b, size_t len) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    bool result = true;
    if (i < len) {
        if (a[i] != b[i]) result = false;
    }
    return result;
}

__device__ bool d_result;

__global__ void matchFile(const char* file_data, size_t file_len, const char* signature, size_t len)
{
	for (int i = 0; i < file_len - len + 1; i++) {
        bool tmp = equal(file_data + i, signature, len);
        if (!tmp) d_result = false;
    }
}

// Convert array of hexadecimal numbers to char*
char* to_hex(uint8_t* data, size_t len) {
  	std::ostringstream oss;
  	oss << std::hex << std::setfill('0');
  	for (size_t i = 0; i < len; i++) {
    	oss << std::setw(2) << (unsigned int)data[i];
  	}
	auto s = oss.str();
  	return &s[0];
}

void runScanner(std::vector<Signature>& signatures, std::vector<InputFile>& inputs)
{
	{
		hipDeviceProp_t prop;
		check_cuda_error(hipGetDeviceProperties(&prop, 0));

		fprintf(stderr, "cuda stats:\n");
		fprintf(stderr, "  # of SMs: %d\n", prop.multiProcessorCount);
		fprintf(stderr, "  global memory: %.2f MB\n", prop.totalGlobalMem / 1024.0 / 1024.0);
		fprintf(stderr, "  shared mem per block: %zu bytes\n", prop.sharedMemPerBlock);
		fprintf(stderr, "  constant memory: %zu bytes\n", prop.totalConstMem);
	}

	/*
		Here, we are creating one stream per file just for demonstration purposes;
		you should change this to fit your own algorithm and/or implementation.
	*/
	std::vector<hipStream_t> streams {};
	streams.resize(inputs.size());

	std::vector<char*> file_bufs {};
	for(size_t i = 0; i < inputs.size(); i++)
	{
		hipStreamCreate(&streams[i]);

		// allocate memory on the device for the file
		char* ptr = 0;
		check_cuda_error(hipMalloc(&ptr, inputs[i].size));
		file_bufs.push_back(ptr);
	}

	// allocate memory for the signatures
	std::vector<char*> sig_bufs {};
	for(size_t i = 0; i < signatures.size(); i++)
	{
		char* ptr = 0;
		check_cuda_error(hipMalloc(&ptr, signatures[i].size));
		hipMemcpy(ptr, signatures[i].data, signatures[i].size, hipMemcpyHostToDevice);
		sig_bufs.push_back(ptr);
	}

	for(size_t file_idx = 0; file_idx < inputs.size(); file_idx++)
	{
		auto input = to_hex(inputs[file_idx].data, inputs[file_idx].size);
		// asynchronously copy the file contents from host memory
		// (the `inputs`) to device memory (file_bufs, which we allocated above)
		hipMemcpyAsync(file_bufs[file_idx], input, inputs[file_idx].size,
			hipMemcpyHostToDevice, streams[file_idx]);    // pass in the stream here to do this async

		for(size_t sig_idx = 0; sig_idx < signatures.size(); sig_idx++)
		{
			// launch the kernel!
			// your job: figure out the optimal dimensions
			int threadsPerBlock = 256;
			auto sz = signatures[sig_idx].size;
			int blocksPerGrid = (sz + threadsPerBlock - 1) / threadsPerBlock;

			bool result = true;
    		hipMemcpyToSymbol(HIP_SYMBOL(d_result), &result, sizeof(bool));
			/*
				This launch happens asynchronously. This means that the CUDA driver returns control
				to our code immediately, without waiting for the kernel to finish. We can then
				run another iteration of this loop to launch more kernels.

				Each operation on a given stream is serialised; in our example here, we launch
				all signatures on the same stream for a file, meaning that, in practice, we get
				a maximum of NUM_INPUTS kernels running concurrently.

				Of course, the hardware can have lower limits; on Compute Capability 8.0, at most
				128 kernels can run concurrently --- subject to resource constraints. This means
				you should *definitely* be doing more work per kernel than in our example!
			*/
			matchFile<<<blocksPerGrid, threadsPerBlock, /* shared memory per block: */ 0, streams[file_idx]>>>(
				file_bufs[file_idx], inputs[file_idx].size,
				sig_bufs[sig_idx], signatures[sig_idx].size);
			
			hipMemcpyFromSymbol(&result, HIP_SYMBOL(d_result), sizeof(bool));

			// example output printing. don't forget to change this!
			if (result) {
				printf("%s: %s\n", inputs[file_idx].name.c_str(), signatures[sig_idx].name.c_str());
			}
		}
	}


	// free the device memory, though this is not strictly necessary
	// (the CUDA driver will clean up when your program exits)
	for(auto buf : file_bufs)
		hipFree(buf);

	for(auto buf : sig_bufs)
		hipFree(buf);

	// clean up streams (again, not strictly necessary)
	for(auto& s : streams)
		hipStreamDestroy(s);
}
